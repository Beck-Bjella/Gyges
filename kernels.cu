#include "hip/hip_runtime.h"
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <cstdio>

// ==================== CONSTANTS ====================

#define MATRIX_DIM 38
#define BIT_36_MASK (1ULL << 36)
#define BIT_37_MASK (1ULL << 37)
#define GOALS_MASK (BIT_36_MASK | BIT_37_MASK)

// Lookup table for intercepts. 0-35 are for twos, 36-71 are for threes.
__constant__ uint64_t ALL_INTERCEPTS[72] {
    // TWOS
    0b00000000000000000000000000000000000000000000000000000001000010ULL,
    0b00000000000000000000000000000000000000000000000000000010000101ULL,
    0b00000000000000000000000000000000000000000000000000000100001010ULL,
    0b00000000000000000000000000000000000000000000000000001000010100ULL,
    0b00000000000000000000000000000000000000000000000000010000101000ULL,
    0b00000000000000000000000000000000000000000000000000100000010000ULL,
    0b00000000000000000000000000000000000000000000000001000010000001ULL,
    0b00000000000000000000000000000000000000000000000010000101000010ULL,
    0b00000000000000000000000000000000000000000000000100001010000100ULL,
    0b00000000000000000000000000000000000000000000001000010100001000ULL,
    0b00000000000000000000000000000000000000000000010000101000010000ULL,
    0b00000000000000000000000000000000000000000000100000010000100000ULL,
    0b00000000000000000000000000000000000000000001000010000001000000ULL,
    0b00000000000000000000000000000000000000000010000101000010000000ULL,
    0b00000000000000000000000000000000000000000100001010000100000000ULL,
    0b00000000000000000000000000000000000000001000010100001000000000ULL,
    0b00000000000000000000000000000000000000010000101000010000000000ULL,
    0b00000000000000000000000000000000000000100000010000100000000000ULL,
    0b00000000000000000000000000000000000001000010000001000000000000ULL,
    0b00000000000000000000000000000000000010000101000010000000000000ULL,
    0b00000000000000000000000000000000000100001010000100000000000000ULL,
    0b00000000000000000000000000000000001000010100001000000000000000ULL,
    0b00000000000000000000000000000000010000101000010000000000000000ULL,
    0b00000000000000000000000000000000100000010000100000000000000000ULL,
    0b00000000000000000000000000000001000010000001000000000000000000ULL,
    0b00000000000000000000000000000010000101000010000000000000000000ULL,
    0b00000000000000000000000000000100001010000100000000000000000000ULL,
    0b00000000000000000000000000001000010100001000000000000000000000ULL,
    0b00000000000000000000000000010000101000010000000000000000000000ULL,
    0b00000000000000000000000000100000010000100000000000000000000000ULL,
    0b00000000000000000000000000000010000001000000000000000000000000ULL,
    0b00000000000000000000000000000101000010000000000000000000000000ULL,
    0b00000000000000000000000000001010000100000000000000000000000000ULL,
    0b00000000000000000000000000010100001000000000000000000000000000ULL,
    0b00000000000000000000000000101000010000000000000000000000000000ULL,
    0b00000000000000000000000000010000100000000000000000000000000000ULL,
    // THREES
    0b0000000000000000000000000000000000000000000000000001000011000110ULL,
    0b0000000000000000000000000000000000000000000000000010000111001101ULL,
    0b0000000000000000000000000000000000000000000000000100001110011011ULL,
    0b0000000000000000000000000000000000000000000000001000011100110110ULL,
    0b0000000000000000000000000000000000000000000000010000111000101100ULL,
    0b0000000000000000000000000000000000000000000000100000110000011000ULL,
    0b0000000000000000000000000000000000000000000001000011000110000011ULL,
    0b0000000000000000000000000000000000000000000010000111001101000111ULL,
    0b0000000000000000000000000000000000000000000100001110011011001110ULL,
    0b0000000000000000000000000000000000000000001000011100110110011100ULL,
    0b0000000000000000000000000000000000000000010000111000101100111000ULL,
    0b0000000000000000000000000000000000000000100000110000011000110000ULL,
    0b0000000000000000000000000000000000000001000011000110000011000001ULL,
    0b0000000000000000000000000000000000000010000111001101000111000010ULL,
    0b0000000000000000000000000000000000000100001110011011001110000100ULL,
    0b0000000000000000000000000000000000001000011100110110011100001000ULL,
    0b0000000000000000000000000000000000010000111000101100111000010000ULL,
    0b0000000000000000000000000000000000100000110000011000110000100000ULL,
    0b0000000000000000000000000000000001000011000110000011000001000000ULL,
    0b0000000000000000000000000000000010000111001101000111000010000000ULL,
    0b0000000000000000000000000000000100001110011011001110000100000000ULL,
    0b0000000000000000000000000000001000011100110110011100001000000000ULL,
    0b0000000000000000000000000000010000111000101100111000010000000000ULL,
    0b0000000000000000000000000000100000110000011000110000100000000000ULL,
    0b0000000000000000000000000000000011000110000011000001000000000000ULL,
    0b0000000000000000000000000000000111001101000111000010000000000000ULL,
    0b0000000000000000000000000000001110011011001110000100000000000000ULL,
    0b0000000000000000000000000000011100110110011100001000000000000000ULL,
    0b0000000000000000000000000000111000101100111000010000000000000000ULL,
    0b0000000000000000000000000000110000011000110000100000000000000000ULL,
    0b0000000000000000000000000000000110000011000001000000000000000000ULL,
    0b0000000000000000000000000000001101000111000010000000000000000000ULL,
    0b0000000000000000000000000000011011001110000100000000000000000000ULL,
    0b0000000000000000000000000000110110011100001000000000000000000000ULL,
    0b0000000000000000000000000000101100111000010000000000000000000000ULL,
    0b0000000000000000000000000000011000110000100000000000000000000000ULL

};

// ==================== HELPER FUNCTIONS ====================

extern "C" __device__ void print_bb(uint64_t bb) {
    for (int i = 0; i < 38; i++) {
        if (bb & ((uint64_t)1 << i)) {
            printf("1 ");

        } else {
            printf("0 ");

        }

    }

    printf("\n");

}

// ==================== BIT STATE HELPERS ====================

// Remove an existing piece type
__device__ void remove_type(uint64_t* state, uint8_t piece_idx) {
    uint8_t type_pos = 38 + (piece_idx * 2);

    // Clear the two bits at type_pos
    uint64_t clear_mask = ~((uint64_t)0b11 << type_pos);
    *state &= clear_mask;

    // Shift higher bits down by 2
    uint64_t higher_bits_mask = ~((uint64_t)0) << (type_pos + 2);
    uint64_t higher_bits = (*state & higher_bits_mask) >> 2;
    *state = (*state & ~higher_bits_mask) | higher_bits;
}

// Add a new piece type
__device__ void add_type(uint64_t* state, uint8_t piece_idx, uint8_t piece_type) {
    uint8_t type_pos = 38 + (piece_idx * 2);

    // Shift higher bits up by 2 to make space
    uint64_t higher_bits_mask = ~((uint64_t)0) << type_pos;
    uint64_t higher_bits = (*state & higher_bits_mask) << 2;
    *state = (*state & ~higher_bits_mask) | higher_bits;

    // Set the new piece type
    *state |= ((uint64_t)piece_type << type_pos);
}

// Change the type data at a specific index
__device__ void set_type_data(uint64_t* state, uint8_t piece_idx, uint8_t piece_type) {
    uint8_t type_pos = 38 + (piece_idx * 2);

    // Mask to clear the two bits at type_pos
    uint64_t clear_mask = ~((uint64_t)0b11 << type_pos);

    // Clear and set the piece type bits at piece_idx
    *state = (*state & clear_mask) | ((uint64_t)piece_type << type_pos);
}

// Get the piece bitboard
__device__ uint64_t get_piece_bb(uint64_t state) {
    return state & (((uint64_t)1 << 38) - 1);
}

// Get the index of the piece at a position
__device__ uint8_t piece_idx(uint64_t state, uint8_t pos) {
    uint64_t mask = ((uint64_t)1 << pos) - 1;
    uint64_t bits_before = get_piece_bb(state) & mask;

    // Use CUDA intrinsic to count set bits
    return __popcll(bits_before);
}

// Get the piece type at a given piece index
__device__ uint8_t piece_type(uint64_t state, uint8_t piece_idx) {
    return (uint8_t)((state >> (38 + (piece_idx * 2))) & 0b11);
}

// Get the piece at a square
// 0 = None, 1 = One, 2 = Two, 3 = Three
__device__ uint8_t piece_at(uint64_t state, uint8_t pos) {
    if ((state & ((uint64_t)1 << pos)) == 0) {
        return 0;
    }

    uint8_t idx = piece_idx(state, pos);
    return piece_type(state, idx);
}

// ==================== BIT STATE ====================

// Get the player to move 
__device__ uint8_t player(uint64_t state) {
    return (uint8_t)(state >> 63);
}

// Make a bounce move
__device__ uint64_t make_bounce_mv(uint64_t state, uint8_t start_pos, uint8_t end_pos) {
    uint64_t new_state = state;

    // Update the piece bitboard
    new_state ^= ((uint64_t)1 << start_pos) | ((uint64_t)1 << end_pos);

    // Starting piece index and type
    uint8_t starting_idx = piece_idx(state, start_pos);
    uint8_t starting_piece = piece_type(state, starting_idx);

    // Remove the piece type at starting index
    remove_type(&new_state, starting_idx);

    // Since we've removed a piece type, adjust the ending index
    uint8_t ending_idx;
    if (start_pos < end_pos) {
        ending_idx = piece_idx(state, end_pos) - 1;
    } else {
        ending_idx = piece_idx(state, end_pos);
    }

    // Add the starting piece type at the new index
    add_type(&new_state, ending_idx, starting_piece);

    return new_state;
}

// Make a drop move
__device__ uint64_t make_drop_mv(uint64_t state, uint8_t start_pos, uint8_t pickup_pos, uint8_t drop_pos) {
    uint64_t new_state = state;

    // Starting piece index and type
    uint8_t starting_idx = piece_idx(state, start_pos);
    uint8_t starting_piece = piece_type(state, starting_idx);

    // Remove the starting piece
    remove_type(&new_state, starting_idx);
    new_state ^= (uint64_t)1 << start_pos;

    // Pickup piece index and type
    uint8_t pickup_idx = piece_idx(new_state, pickup_pos);
    uint8_t pickup_piece = piece_type(new_state, pickup_idx);

    // Set the pickup piece's type to the starting piece's type
    set_type_data(&new_state, pickup_idx, starting_piece);

    // Drop piece index
    uint8_t drop_idx = piece_idx(new_state, drop_pos);

    // Add the pickup piece type at the drop index
    add_type(&new_state, drop_idx, pickup_piece);

    // Update the piece bitboard
    new_state ^= (uint64_t)1 << drop_pos;

    return new_state;
}

// ==================== HELPERS ====================

extern "C" __device__ uint64_t reach(
    uint64_t piece_bb,
    uint8_t piece_type,
    uint64_t piece_pos,

    // Lookup tables
    const uint64_t* __restrict__ one_reach,
    const uint64_t* __restrict__ two_reach,
    const uint64_t* __restrict__ three_reach
) {
    uint64_t reach;
    uint64_t intercepts;
    if (piece_type == 1) {
        reach = one_reach[piece_pos];
        
    } else if (piece_type == 2) {
        intercepts = piece_bb & ALL_INTERCEPTS[piece_pos];
        reach = two_reach[(piece_pos * 29) + (intercepts % 29)];
        
    } else if (piece_type == 3) {
        intercepts = piece_bb & ALL_INTERCEPTS[piece_pos + 36];
        reach = three_reach[(piece_pos * 11007) + (intercepts % 11007)];

    }

    // Mask reach
    uint64_t masked_reach = reach & (piece_bb | GOALS_MASK);

    return masked_reach;

}

// ==================== BLOCKING KERNELS ======================

// Adj matrix vairant
extern "C" __global__ void adj_kernel(
    uint64_t* init_state, // Starting state
    uint8_t* move_data,   // Move data
    float* routes,        // Output 

    // Lookup tables
    const uint64_t* __restrict__ one_reach,
    const uint64_t* __restrict__ two_reach,
    const uint64_t* __restrict__ three_reach

) {
    uint64_t matrix_id = blockIdx.x;       // Each block processes one matrix
    uint64_t thread_id = threadIdx.x;      // Each thread processes one row

    // Init shared memory
    __shared__ uint64_t adj_matrix[38];
    __shared__ uint64_t result_matrix[38];
    __shared__ uint64_t shared_new_state;

    if (thread_id == 0) {
        uint8_t start_pos = move_data[(matrix_id * 3)];
        uint8_t pickup_pos = move_data[(matrix_id * 3) + 1];
        uint8_t end_pos = move_data[(matrix_id * 3) + 2];

        uint64_t new_state = (end_pos == 100)
            ? make_bounce_mv(init_state[0], start_pos, pickup_pos)
            : make_drop_mv(init_state[0], start_pos, pickup_pos, end_pos);

        shared_new_state = new_state;

    }

    __syncthreads(); // Sync threads

    // Step 1: Generate Adj Matrix
    uint8_t piece_type = piece_at(shared_new_state, thread_id);
    if (piece_type != 0) {
        uint64_t piece_bb = get_piece_bb(shared_new_state);

        adj_matrix[thread_id] = reach(
            piece_bb,
            piece_type,
            thread_id,
            one_reach,
            two_reach,
            three_reach
        );

    } else {
        adj_matrix[thread_id] = 0;

    }
    
    result_matrix[thread_id] = adj_matrix[thread_id];

    __syncthreads(); // Sync threads

    // Step 2: Bitwise Matrix Multiplication
    for (uint64_t exp = 0; exp < 8; exp++) {
        // Each thread computes its rows result
        uint64_t row_result = 0;
        for (uint64_t k = 0; k < 38; k++) {
            row_result |= adj_matrix[k] * ((result_matrix[thread_id] >> k) & 1ULL);

        }

        // Store result
        result_matrix[thread_id] |= row_result;

        __syncthreads(); // Sync before the next power calculation

    }

    // Step 3: Validate Routes
    if (thread_id == 0) {
        for (int row = 30; row < 36; row++) {
            if (result_matrix[row] & BIT_36_MASK) {
                routes[matrix_id] = 1.0; // There is a path to the goal -> did not block the threat
                return;

            }

        }

        routes[matrix_id] = 0.0; // No path to the goal -> blocked the threat

    }

}

// Wavefront variant
extern "C" __global__ void wavefront_kernel(
    uint64_t* init_state, // Starting state
    uint8_t* move_data,   // Move data
    float* routes,        // Output 

    // Lookup tables
    const uint64_t* __restrict__ one_reach,
    const uint64_t* __restrict__ two_reach,
    const uint64_t* __restrict__ three_reach

) {
    uint64_t block_id = blockIdx.x;       // Each block processes one matrix
    uint64_t thread_id = threadIdx.x;     // Each thread processes one row

    // Shared memory
    __shared__ uint64_t current_frontier; // Current wavefront of nodes
    __shared__ uint64_t next_frontier;    // Next wavefront of nodes
    __shared__ uint64_t reached;          // Reached nodes

    __shared__ uint64_t shared_state;
    __shared__ uint64_t shared_bb;

    if (thread_id == 0) {
        // Frontiers
        current_frontier = 0ULL;
        current_frontier |= (1ULL << 31) | (1ULL << 33); // NEED TO CHANGE

        next_frontier = 0ULL;

        reached = current_frontier;
        
        // Shared state
        uint8_t start_pos = move_data[(block_id * 3)];
        uint8_t pickup_pos = move_data[(block_id * 3) + 1];
        uint8_t end_pos = move_data[(block_id * 3) + 2];

        uint64_t new_state = (end_pos == 100)
            ? make_bounce_mv(init_state[0], start_pos, pickup_pos)
            : make_drop_mv(init_state[0], start_pos, pickup_pos, end_pos);

        shared_state = new_state;
        shared_bb = get_piece_bb(new_state);

    }

    __syncthreads(); // Sync threads

    for (int depth = 0; depth < 8; depth++) {
        // Exit condition
        if (current_frontier == 0ULL || current_frontier & BIT_36_MASK) {
            break;
        }

        if ((1ULL << thread_id) & current_frontier) {
            uint8_t piece_type = piece_at(shared_state, thread_id);
            if (piece_type != 0) {
                uint64_t reachable = reach(
                    shared_bb,
                    piece_type,
                    thread_id,
                    one_reach,
                    two_reach,
                    three_reach
                );

                // Remove already reached positions
                reachable &= ~reached;

                // Update next frontier
                atomicOr(&next_frontier, reachable);

            }

        }

        __syncthreads(); // Sync threads

        // Swap frontiers
        if (thread_id == 0) {
            current_frontier = next_frontier;
            next_frontier = 0ULL;

            atomicOr(&reached, current_frontier);

        }

        __syncthreads(); // Sync threads

    }

    // Store result
    if (thread_id == 0) {
        routes[block_id] = (current_frontier & BIT_36_MASK) ? 1.0f : 0.0f;

    }

}

// ==============================================================
// ==============================================================
// =======================   NEW STUFF  =========================  
// ==============================================================
// ==============================================================

// Remove an existing piece type
__device__ uint64_t new_remove_type(uint64_t state, uint8_t piece_idx) {
    uint64_t new_state = state;

    uint8_t type_pos = 38 + (piece_idx * 2);

    // Clear the two bits at type_pos
    uint64_t clear_mask = ~((uint64_t)0b11 << type_pos);
    new_state &= clear_mask;

    // Shift higher bits down by 2
    uint64_t higher_bits_mask = ~((uint64_t)0) << (type_pos + 2);
    uint64_t higher_bits = (new_state & higher_bits_mask) >> 2;
    new_state = (new_state & ~higher_bits_mask) | higher_bits;

    return new_state;

}

__device__ uint64_t remove_piece(uint64_t state, uint8_t pos) {
    uint64_t new_state = state;

    // Clear the bit at pos
    new_state &= ~((uint64_t)1 << pos);

    // Remove the piece type
    new_state = new_remove_type(new_state, piece_idx(state, pos));

    return new_state;

}

// ==============================================================

#define MAX_STACK_SIZE 1000

struct OnePath {
    uint64_t backtrack_bb;
    uint8_t p1;
    uint8_t p2;
};

struct TwoPath {
    uint64_t backtrack_bb;
    uint8_t p1;
    uint8_t p2;
    uint8_t p3;
};

struct ThreePath {
    uint64_t backtrack_bb;
    uint8_t p1;
    uint8_t p2;
    uint8_t p3;
    uint8_t p4;
};

struct StackData {
    uint64_t banned_bb;
    uint64_t backtrack_bb;
    uint32_t active_line_idx;
    uint8_t current_pos;
    uint8_t current_piece;

};

struct GenRequest {
    uint64_t state;
    uint64_t active_bb;
    uint8_t flag;

};

struct GenResult {
    uint64_t end_positions[6];
    uint64_t pickup_positions[6];
    uint64_t drop_positions;
    
};

// ==============================================================

// Pushs to a stach w/o any overflow handling
__device__ void push(StackData* stack, uint32_t* stack_height, uint32_t block_id, uint32_t type, StackData data) {
    uint32_t current_height = atomicAdd(stack_height, 1); // Get the position to push to
    stack[(block_id * MAX_STACK_SIZE * 3) + (type * MAX_STACK_SIZE) + current_height] = data;

}

// Pops from a stack w/o any underflow handling
__device__ StackData pop(StackData* stack, uint32_t* stack_height, uint32_t block_id, uint32_t type) {
    uint32_t current_height = atomicSub(stack_height, 1) - 1; // Get the position to pop from
    return stack[(block_id * MAX_STACK_SIZE * 3) + (type * MAX_STACK_SIZE) + current_height];

}

// ==============================================================

// Lookup tables
__device__ OnePath* one_paths;
__device__ TwoPath* two_paths;
__device__ ThreePath* three_paths;
__device__ uint16_t* one_path_lists;
__device__ uint16_t* two_path_lists;
__device__ uint16_t* three_path_lists;
__device__ uint8_t* one_map;
__device__ uint16_t* two_map;
__device__ uint16_t* three_map;

// ==============================================================

__device__ void process_one(
    StackData* stack, 
    uint32_t* one_stack_height, 
    uint32_t* two_stack_height,
    uint32_t* three_stack_height,
    StackData current_data,
    uint64_t current_state,
    uint64_t end_positions,
    uint64_t* local_end_positions, 
    uint64_t* local_pickup_positions,
    uint64_t block_id
) {
    uint16_t path_list_idx = one_map[current_data.current_pos];
    uint16_t path_list_len = one_path_lists[(path_list_idx * 5) + 4];

    for (int i = 0; i < path_list_len; i++) {
        uint16_t path_idx = one_path_lists[(path_list_idx * 5) + i];
        OnePath path = one_paths[path_idx];

        uint8_t end_pos = path.p2;
        uint64_t end_bit = (uint64_t)1 << end_pos;

        uint64_t end_pos_banned = (current_data.banned_bb | end_positions) & end_bit;
        uint64_t backtrack_conflict = current_data.backtrack_bb & path.backtrack_bb; 
        bool valid_player = !(((end_bit & BIT_36_MASK) && 0 == 0) || ((end_bit & BIT_37_MASK) && 0 == 1));

        if (backtrack_conflict || end_pos_banned || !valid_player) {
            continue;

        }           
            
        uint64_t end_piece = piece_at(current_state, end_pos);
        bool is_empty = (end_piece == 0);

        if (is_empty) {
            *local_end_positions |= end_bit;

        } else {
            uint64_t new_banned_bb = current_data.banned_bb ^ end_bit;
            uint64_t new_backtrack_bb = current_data.backtrack_bb ^ path.backtrack_bb;
            
            *local_pickup_positions |= end_bit;

            StackData new_data = {
                new_banned_bb,
                new_backtrack_bb,
                current_data.active_line_idx,
                end_pos,
                (uint8_t)end_piece,
            };

            int stack_type = end_piece - 1;
            push(
                stack, 
                (stack_type == 0) ? one_stack_height : (stack_type == 1) ? two_stack_height : three_stack_height, 
                block_id, 
                stack_type, 
                new_data
            );

        }
            
    }

}

__device__ void process_two(
    StackData* stack, 
    uint32_t* one_stack_height, 
    uint32_t* two_stack_height,
    uint32_t* three_stack_height,
    StackData current_data, 
    uint64_t current_state,
    uint64_t end_positions,
    uint64_t* local_end_positions, 
    uint64_t* local_pickup_positions,
    uint64_t block_id
) {
    uint64_t intercept_bb = get_piece_bb(current_state) & ALL_INTERCEPTS[current_data.current_pos];

    uint16_t path_list_idx = two_map[(current_data.current_pos * 29) + (intercept_bb % 29)];
    uint16_t path_list_len = two_path_lists[(path_list_idx * 13) + 12];

    for (int i = 0; i < path_list_len; i++) {
        uint16_t path_idx = two_path_lists[(path_list_idx * 13) + i];
        TwoPath path = two_paths[path_idx];

        uint8_t end_pos = path.p3;
        uint64_t end_bit = (uint64_t)1 << end_pos;

        uint64_t end_pos_banned = (current_data.banned_bb | end_positions) & end_bit;
        uint64_t backtrack_conflict = current_data.backtrack_bb & path.backtrack_bb; 
        bool valid_player = !(((end_bit & BIT_36_MASK) && 0 == 0) || ((end_bit & BIT_37_MASK) && 0 == 1));

        if (backtrack_conflict || end_pos_banned || !valid_player) {
            continue;

        }           
            
        uint64_t end_piece = piece_at(current_state, end_pos);
        bool is_empty = (end_piece == 0);

        if (is_empty) {
            *local_end_positions |= end_bit;

        } else {
            uint64_t new_banned_bb = current_data.banned_bb ^ end_bit;
            uint64_t new_backtrack_bb = current_data.backtrack_bb ^ path.backtrack_bb;
            
            *local_pickup_positions |= end_bit;

            StackData new_data = {
                new_banned_bb,
                new_backtrack_bb,
                current_data.active_line_idx,
                end_pos,
                (uint8_t)end_piece,
            };

            int stack_type = end_piece - 1;
            push(
                stack, 
                (stack_type == 0) ? one_stack_height : (stack_type == 1) ? two_stack_height : three_stack_height, 
                block_id, 
                stack_type, 
                new_data
            );

        }

    }

}

__device__ void process_three(
    StackData* stack, 
    uint32_t* one_stack_height, 
    uint32_t* two_stack_height,
    uint32_t* three_stack_height,
    StackData current_data, 
    uint64_t current_state,
    uint64_t end_positions,
    uint64_t* local_end_positions, 
    uint64_t* local_pickup_positions,
    uint64_t block_id
) {
    uint64_t intercept_bb = get_piece_bb(current_state) & ALL_INTERCEPTS[current_data.current_pos + 36];

    uint16_t path_list_idx = three_map[(current_data.current_pos * 11007) + (intercept_bb % 11007)];
    uint16_t path_list_len = three_path_lists[(path_list_idx * 36) + 35];

    for (int i = 0; i < path_list_len; i++) {
        uint16_t path_idx = three_path_lists[(path_list_idx * 36) + i];
        ThreePath path = three_paths[path_idx];

        uint8_t end_pos = path.p4;
        uint64_t end_bit = (uint64_t)1 << end_pos;

        uint64_t end_pos_banned = (current_data.banned_bb | end_positions) & end_bit;
        uint64_t backtrack_conflict = current_data.backtrack_bb & path.backtrack_bb; 
        bool valid_player = !(((end_bit & BIT_36_MASK) && 0 == 0) || ((end_bit & BIT_37_MASK) && 0 == 1));

        if (backtrack_conflict || end_pos_banned || !valid_player) {
            continue;

        }           
            
        uint64_t end_piece = piece_at(current_state, end_pos);
        bool is_empty = (end_piece == 0);

        if (is_empty) {
            *local_end_positions |= end_bit;

        } else {
            uint64_t new_banned_bb = current_data.banned_bb ^ end_bit;
            uint64_t new_backtrack_bb = current_data.backtrack_bb ^ path.backtrack_bb;
            
            *local_pickup_positions |= end_bit;

            StackData new_data = {
                new_banned_bb,
                new_backtrack_bb,
                current_data.active_line_idx,
                end_pos,
                (uint8_t)end_piece,
            };

            int stack_type = end_piece - 1;
            push(
                stack, 
                (stack_type == 0) ? one_stack_height : (stack_type == 1) ? two_stack_height : three_stack_height, 
                block_id, 
                stack_type, 
                new_data
            );

        }

    } 

}

extern "C" __global__ void gen_kernel(
    const GenRequest* __restrict__ in_data,
    GenResult* out_data,  
    StackData* stack
    
) {
    uint32_t block_id = blockIdx.x;       // Each block processes one generation request
    uint32_t thread_id = threadIdx.x;     // 
    uint32_t warp_id = threadIdx.x / 32;  //
    uint32_t lane_id = threadIdx.x % 32;  // 

    // Init Shared Memory
    __shared__ uint32_t one_stack_height;       // Stack height for ones
    __shared__ uint32_t two_stack_height;       // Stack height for twos
    __shared__ uint32_t three_stack_height;     // Stack height for threes
    __shared__ uint64_t init_state;             // Initial state
    __shared__ uint64_t end_positions[6];       // End positions
    __shared__ uint64_t pickup_positions[6];    // Pickup positions
    __shared__ uint64_t drop_positions;         // Drop positions
    if (thread_id == 0) {
        one_stack_height = 0;
        two_stack_height = 0;
        three_stack_height = 0;
        init_state = in_data[block_id].state;

        for (int i = 0; i < 6; i++) {
            end_positions[i] = 0ULL;
            pickup_positions[i] = 0ULL;

        }
        drop_positions = (~init_state & 0b111111111111111111111111111111111111ULL);

    }

    __syncthreads(); // Sync threads
    
    // Setup
    __shared__ uint64_t starting_states[6]; // Starting states for active line
    uint64_t start_bb = in_data[block_id].active_bb;
    if (((1ULL << thread_id) & start_bb) && thread_id < 6) {
        uint64_t new_state = remove_piece(init_state, thread_id);
        starting_states[thread_id] = new_state; // WRONG INDEX -> ONLY WORKS WHEN STARTING LINE IS ON THE FIRST ROW

        // Create init stack data
        uint8_t piece = piece_at(init_state, thread_id);
        StackData data = {
            0ULL,
            0ULL,
            thread_id, // WRONG INDEX -> ONLY WORKS WHEN STARTING LINE IS ON THE FIRST ROW
            (uint8_t)thread_id, 
            piece,
        };

        // Push to stack
        int stack_type = piece - 1;
        push(
            stack, 
            (stack_type == 0) ? &one_stack_height : (stack_type == 1) ? &two_stack_height : &three_stack_height, 
            block_id, 
            stack_type, 
            data
        );

    }
    
    __syncthreads(); // Sync threads

    // MAIN PROCESSING LOOP
    while (true) {
        if (warp_id == 0 && lane_id < one_stack_height) { // ONES -> warp 0
            StackData current_data = pop(stack, &one_stack_height, block_id, 0);
            uint64_t current_state = starting_states[current_data.active_line_idx];

            uint64_t local_end_positions = 0ULL;
            uint64_t local_pickup_positions = 0ULL;

            process_one(
                stack,
                &one_stack_height,
                &two_stack_height,
                &three_stack_height,
                current_data,
                current_state,
                end_positions[current_data.active_line_idx],
                &local_end_positions,
                &local_pickup_positions,
                block_id
            );

            atomicOr(&end_positions[current_data.active_line_idx], local_end_positions);
            atomicOr(&pickup_positions[current_data.active_line_idx], local_pickup_positions);

        } else if (warp_id == 1 && lane_id < two_stack_height) { // TWOS -> warp 1
            StackData current_data = pop(stack, &two_stack_height, block_id, 1);
            uint64_t current_state = starting_states[current_data.active_line_idx];

            uint64_t local_end_positions = 0ULL;
            uint64_t local_pickup_positions = 0ULL;

            process_two(
                stack,
                &one_stack_height,
                &two_stack_height,
                &three_stack_height,
                current_data,
                current_state,
                end_positions[current_data.active_line_idx],
                &local_end_positions,
                &local_pickup_positions,
                block_id
            );

            atomicOr(&end_positions[current_data.active_line_idx], local_end_positions);
            atomicOr(&pickup_positions[current_data.active_line_idx], local_pickup_positions);

        } else if (warp_id == 2 && lane_id < three_stack_height) { // THREES -> warp 2
            StackData current_data = pop(stack, &three_stack_height, block_id, 2);
            uint64_t current_state = starting_states[current_data.active_line_idx];

            uint64_t local_end_positions = 0ULL;
            uint64_t local_pickup_positions = 0ULL;

            process_three(
                stack,
                &one_stack_height,
                &two_stack_height,
                &three_stack_height,
                current_data,
                current_state,
                end_positions[current_data.active_line_idx],
                &local_end_positions,
                &local_pickup_positions,
                block_id
            );

            atomicOr(&end_positions[current_data.active_line_idx], local_end_positions);
            atomicOr(&pickup_positions[current_data.active_line_idx], local_pickup_positions);

        }

        __syncthreads(); // Sync threads

        // Exit condition
        if (one_stack_height == 0 && two_stack_height == 0 && three_stack_height == 0) {
            break;

        }

    }

    __syncthreads(); // Sync threads

    // Store results
    if (thread_id == 0) {
        GenResult result;
        for (int i = 0; i < 6; i++) {
            result.end_positions[i] = end_positions[i];
            result.pickup_positions[i] = pickup_positions[i];

        }
        result.drop_positions = drop_positions;

        out_data[block_id] = result;

    }

}

// ==============================================================


// extern "C" __global__ void gen_kernel(
//     const GenRequest* __restrict__ in_data,
//     GenResult* out_data,   
//     StackData* stack
    
// ) {
//     uint32_t block_id = blockIdx.x;       // Each block processes one generation request
//     uint32_t thread_id = threadIdx.x;     // Each thread processes one postion

//     // Init Shared Memory
//     __shared__ uint32_t stack_height;           // Stack height
//     __shared__ uint64_t init_state;             // Initial state
//     __shared__ uint64_t end_positions[6];       // End positions
//     __shared__ uint64_t pickup_positions[6];    // Pickup positions
//     __shared__ uint64_t drop_positions;         // Drop positions
//     if (thread_id == 0) {
//         stack_height = 0;
//         init_state = in_data[block_id].state;

//         for (int i = 0; i < 6; i++) {
//             end_positions[i] = 0ULL;
//             pickup_positions[i] = 0ULL;

//         }
//         drop_positions = (~init_state & 0b111111111111111111111111111111111111ULL);

//     }

//     __syncthreads(); // Sync threads
    
//     // Setup
//     __shared__ uint64_t starting_states[6]; // Starting states
//     uint64_t start_bb = in_data[block_id].active_bb;
//     if ((1ULL << thread_id) & start_bb) {
//         // Save starting info to shared memory
//         uint64_t new_state = remove_piece(init_state, thread_id);
//         starting_states[thread_id] = new_state; // WRONG INDEX -> ONLY WORKS WHEN STARTING LINE IS ON THE FIRST ROW

//         // Store starting data into stack
//         StackData data = {
//             0ULL,
//             0ULL,
//             thread_id, // WRONG INDEX -> ONLY WORKS WHEN STARTING LINE IS ON THE FIRST ROW
//             (uint8_t)thread_id, 
//             piece_at(init_state, thread_id),
//         };

//         push(stack, &stack_height, block_id, data);

//     }

//     __syncthreads(); // Sync threads

//     // TESTING PURPOSES
//     uint16_t player = 0;    

//     // MAIN PROCESSING LOOP
//     while (true) {
//         if (thread_id < stack_height) {
//             StackData current_data = pop(stack, &stack_height, block_id);
//             uint64_t current_state = starting_states[current_data.active_line_idx];

//             uint64_t local_end_positions = 0ULL;
//             uint64_t local_pickup_positions = 0ULL;

//             if (current_data.current_piece == 1) { // ONES
//                 uint16_t path_list_idx = one_map[current_data.current_pos];
//                 uint16_t path_list_len = one_path_lists[(path_list_idx * 5) + 4];

//                 for (int i = 0; i < path_list_len; i++) {
//                     uint16_t path_idx = one_path_lists[(path_list_idx * 5) + i];
//                     OnePath path = one_paths[path_idx];

//                     if (current_data.backtrack_bb & path.backtrack_bb) {
//                         continue;

//                     }

//                     uint8_t end_pos = path.p2;
//                     uint64_t end_bit = (uint64_t)1 << end_pos;

//                     if (end_positions[current_data.active_line_idx] & end_bit) {
//                         continue;

//                     }

//                     if (end_bit & BIT_36_MASK) {
//                         if (player == 0) {
//                             continue;
//                         }
//                         local_end_positions |= end_bit;

//                     } else if (end_bit & BIT_37_MASK) {
//                         if (player == 1) {
//                             continue;
//                         }
//                         local_end_positions |= end_bit;

//                     }

//                     uint64_t end_piece = piece_at(current_state, end_pos);
//                     if (end_piece != 0) {
//                         if ((current_data.banned_bb & end_bit) == 0) {
//                             uint64_t new_banned_bb = current_data.banned_bb ^ end_bit;
//                             uint64_t new_backtrack_bb = current_data.backtrack_bb ^ path.backtrack_bb;

//                             local_pickup_positions |= end_bit;

//                             StackData new_data = {
//                                 new_banned_bb,
//                                 new_backtrack_bb,
//                                 current_data.active_line_idx,
//                                 end_pos,
//                                 (uint8_t)end_piece,
//                             };

//                             push(stack, &stack_height, block_id, new_data);

//                         }

//                     } else {
//                         local_end_positions |= end_bit;

//                     }
                        
//                 }

//             } else if (current_data.current_piece == 2) { // TWOS
//                 uint64_t intercept_bb = get_piece_bb(current_state) & ALL_INTERCEPTS[current_data.current_pos];

//                 uint16_t path_list_idx = two_map[(current_data.current_pos * 29) + (intercept_bb % 29)];
//                 uint16_t path_list_len = two_path_lists[(path_list_idx * 13) + 12];

//                 for (int i = 0; i < path_list_len; i++) {
//                     uint16_t path_idx = two_path_lists[(path_list_idx * 13) + i];
//                     TwoPath path = two_paths[path_idx];

//                     if (current_data.backtrack_bb & path.backtrack_bb) {
//                         continue;

//                     }

//                     uint8_t end_pos = path.p3;
//                     uint64_t end_bit = (uint64_t)1 << end_pos;

//                     if (end_positions[current_data.active_line_idx] & end_bit) {
//                         continue;

//                     }

//                     if (end_bit & BIT_36_MASK) {
//                         if (player == 0) {
//                             continue;
//                         }
//                         local_end_positions |= end_bit;
                        
//                     } else if (end_bit & BIT_37_MASK) {
//                         if (player == 1) {
//                             continue;
//                         }
//                         local_end_positions |= end_bit;

//                     }

//                     uint64_t end_piece = piece_at(current_state, end_pos);
//                     if (end_piece != 0) {
//                         if ((current_data.banned_bb & end_bit) == 0) {
//                             uint64_t new_banned_bb = current_data.banned_bb ^ end_bit;
//                             uint64_t new_backtrack_bb = current_data.backtrack_bb ^ path.backtrack_bb;

//                             local_pickup_positions |= end_bit;

//                             StackData new_data = {
//                                 new_banned_bb,
//                                 new_backtrack_bb,
//                                 current_data.active_line_idx,
//                                 end_pos,
//                                 (uint8_t)end_piece,
//                             };

//                             push(stack, &stack_height, block_id, new_data);

//                         }

//                     } else {
//                         local_end_positions |= end_bit;

//                     }

//                 }

//             } else if (current_data.current_piece == 3) {
//                 uint64_t intercept_bb = get_piece_bb(current_state) & ALL_INTERCEPTS[current_data.current_pos + 36];

//                 uint16_t path_list_idx = three_map[(current_data.current_pos * 11007) + (intercept_bb % 11007)];
//                 uint16_t path_list_len = three_path_lists[(path_list_idx * 36) + 35];

//                 for (int i = 0; i < path_list_len; i++) {
//                     uint16_t path_idx = three_path_lists[(path_list_idx * 36) + i];
//                     ThreePath path = three_paths[path_idx];

//                     if (current_data.backtrack_bb & path.backtrack_bb) {
//                         continue;

//                     }

//                     uint8_t end_pos = path.p4;
//                     uint64_t end_bit = (uint64_t)1 << end_pos;

//                     if (end_positions[current_data.active_line_idx] & end_bit) {
//                         continue;

//                     }

//                     if (end_bit & BIT_36_MASK) {
//                         if (player == 0) {
//                             continue;
//                         }
//                         local_end_positions |= end_bit;
       
//                     } else if (end_bit & BIT_37_MASK) {
//                         if (player == 1) {
//                             continue;
//                         }
//                         local_end_positions |= end_bit;
               
//                     }
                        
//                     uint64_t end_piece = piece_at(current_state, end_pos);
//                     if (end_piece != 0) {
//                         if ((current_data.banned_bb & end_bit) == 0) {
//                             uint64_t new_banned_bb = current_data.banned_bb ^ end_bit;
//                             uint64_t new_backtrack_bb = current_data.backtrack_bb ^ path.backtrack_bb;
                            
//                             local_pickup_positions |= end_bit;

//                             StackData new_data = {
//                                 new_banned_bb,
//                                 new_backtrack_bb,
//                                 current_data.active_line_idx,
//                                 end_pos,
//                                 (uint8_t)end_piece,
//                             };

//                             push(stack, &stack_height, block_id, new_data);
                            
//                         }

//                     } else {
//                         local_end_positions |= end_bit;

//                     }

//                 } 

//             }

//             // Sync Data
//             if (current_data.current_piece != 0) {
//                 atomicOr(&end_positions[current_data.active_line_idx], local_end_positions);
//                 atomicOr(&pickup_positions[current_data.active_line_idx], local_pickup_positions);

//             }
            
//         }

//         __syncthreads(); // Sync threads

//         // Exit condition
//         if (stack_height == 0) {
//             break;

//         }

//     }

//     __syncthreads(); // Sync threads

//     // Store results
//     if (thread_id == 0) {
//         GenResult result;
//         for (int i = 0; i < 6; i++) {
//             result.end_positions[i] = end_positions[i];
//             result.pickup_positions[i] = pickup_positions[i];

//         }
//         result.drop_positions = drop_positions;

//         out_data[block_id] = result;

//     }

// }
